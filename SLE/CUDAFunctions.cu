#include "hip/hip_runtime.h"
//
//  CUDAfunctions.cu
//  SLE
//
//  Created by Henry Jackson on 19/06/2015.
//  Copyright (c) 2015 n/a. All rights reserved.
//

#include "CUDAfunctions.cuh"

/*--- Reverse Functions ---*/

__global__ void updateMatrixReverseGPU(int start, int end,
	                                double offset,
									cpx* inMat, cpx* outMat,
									double* dt, double* shift,
									int rows, int cols)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	if (Row < rows && Col < cols){
		updateValueReverse(start, end, offset, inMat[Row*cols + Col], outMat[Row*cols + Col], dt, shift);
	}
}

__device__ cpx slitMap(int index, cpx inValue, double* dt, double* shifts){
	return sqrt(4 * dt[index] - inValue*inValue)*cpx(0, 1) + shifts[index];
}

__device__ void updateValueReverse(int start, int end,
	double offset,
	cpx inValue,
	cpx& outValue,
	double* dt, double* shift)
{
	outValue = slitMap(end - 1, inValue - offset, dt, shift);
	for (int i = end - 2; i >= start; --i) {
		outValue = slitMap(i, outValue, dt, shift);
	}
}

/*--- Forward Functions ---*/

__global__ void updateMatrixForwardGPU(int start, int end,
	cpx* inMat, cpx* outMat,
	double* dt, double* shift,
	int rows, int cols)
{
	int Row = blockIdx.y*blockDim.y + threadIdx.y;
	int Col = blockIdx.x*blockDim.x + threadIdx.x;

	if (Row < rows && Col < cols){
		updateValueForward(start, end, inMat[Row*cols + Col], outMat[Row*cols + Col], dt, shift);
	}
}

__device__ void updateValueForward(int start, int end,
	cpx inValue,
	cpx& outValue,
	double* dt, double* shift)
{
	outValue = slitMapInverse(start, inValue, dt, shift);
	for (int i = start + 1; i < end; ++i) {
		outValue = slitMapInverse(i, outValue, dt, shift);
	}
}

__device__ cpx slitMapInverse(int index, cpx inValue, double* dt, double* shifts){
	cpx outValue = sqrt(-(inValue - shifts[index])*(inValue - shifts[index]) - 4 * dt[index]) * cpx(0, 1);
	return outValue;
}